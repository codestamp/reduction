#include "timinggpu.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

struct PrivateTimingGPU {
	hipEvent_t start;
	hipEvent_t stop; 
};

//default constructor
TimingGPU::TimingGPU() { privateTimingGPU = new PrivateTimingGPU(); }

//default destructor
TimingGPU::~TimingGPU() {}

void TimingGPU::StartCounter() {
	hipEventCreate(&((*privateTimingGPU).start));
	hipEventCreate(&((*privateTimingGPU).stop));
}

void TimingGPU::StartCounterFlags() {
	int eventFlags=hipEventBlockingSync;

	hipEventCreateWithFlags(&((*privateTimingGPU).start),eventFlags);
	hipEventCreateWithFlags(&((*privateTimingGPU).stop),eventFlags);
	hipEventRecord((*privateTimingGPU).start,0);
}

//gets the counter in milliseconds
float TimingGPU::GetCounter() {
	float time;
	hipEventRecord((*privateTimingGPU).stop,0);
	hipEventSynchronize((*privateTimingGPU).stop);
	hipEventElapsedTime(&time,(*privateTimingGPU).start,(*privateTimingGPU).stop);
	return time;
}



