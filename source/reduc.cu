#include "hip/hip_runtime.h"
/*
 * reduc.cu
 *
 *  Created on: Jun 12, 2017
 *      Author: Munesh Singh
 */
#include "utils.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define ARRSZ (1 << 25)
#define BLOCK_SIZE 1024

using namespace std;

__global__
void reduce0(int *g_idata,int *g_odata)
{
	extern __shared__ int sdata[];

	//each thread loads one element from global to shared memory
	unsigned int tid=threadIdx.x;
	unsigned int i=threadIdx.x+blockIdx.x*blockDim.x;

	sdata[tid]=g_idata[i];
	__syncthreads();


	//do reduction in shared memory
	for(unsigned int s=1;s<blockDim.x;s*=2) {
		if((tid%(2*s))==0) {
			sdata[tid] += sdata[tid+s];
		}
		__syncthreads();
	}

	//write the result for this block to global mem
	if(tid==0)
		g_odata[blockIdx.x]=sdata[0];

}

int main() {
	srand(time(0));
	//host allocations
	int *h_idata, *h_odata;

	//allocating and initializing host memory
	h_idata=(int*)malloc(ARRSZ * sizeof(int));
	for(int i=0;i<ARRSZ;i++)
		h_idata[i]=randomNumber();

	for(int i=0;i<10;i++)
			cout << h_idata[i] << endl;

	//allocating device memories
	int *g_idata, *g_odata;
	gpuErrchk(hipMalloc((void**)&g_idata,sizeof(int)*ARRSZ));


	int GRID_SIZE=divup(ARRSZ,BLOCK_SIZE);

	cout << "GRID_SIZE= " << GRID_SIZE << endl;
	gpuErrchk(hipMalloc((void**)&g_odata,sizeof(int)*GRID_SIZE));
	h_odata=(int*)malloc(GRID_SIZE * sizeof(int));
	//gpuErrchk(hipMemset(g_odata,0,sizeof(int)*GRID_SIZE));

	gpuErrchk(hipMemcpy(g_idata,h_idata,sizeof(int)*ARRSZ,hipMemcpyHostToDevice));

	reduce0<<<GRID_SIZE,BLOCK_SIZE, sizeof(int)*BLOCK_SIZE>>>(g_idata,g_odata);

	gpuErrchk(hipMemcpy(h_odata,g_odata,sizeof(int)*GRID_SIZE,hipMemcpyDeviceToHost));

	for(int i=0;i<GRID_SIZE;i++)
		cout << h_odata[i] << endl;

	//Serial reduce
	int sum=0;
	for(int i=0;i<ARRSZ;i++)
		sum+=h_idata[i];

	cout << "Serial sum = " << sum << endl;

	sum=0;
	for(int i=0;i<GRID_SIZE;i++)
			sum+=h_odata[i];

	cout << "Parallel sum = " << sum << endl;


	free(h_idata);
	free(h_odata);
	hipFree(g_idata);
	hipFree(g_odata);

	return 0;
}
