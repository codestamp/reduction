#include "hip/hip_runtime.h"
/*
 * reduc.cu
 *
 *  Created on: Jun 12, 2017
 *      Author: Munesh Singh
 */
#include "utils.cuh"
#include "gputimer.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define ARRSZ (1 << 22)
#define BLOCK_SIZE 128

using namespace std;

__global__
void reduce3(int *g_idata,int *g_odata)
{
	extern __shared__ int sdata[];

	//each thread loads one element from global to shared memory
	unsigned int tid=threadIdx.x;
	unsigned int i=threadIdx.x+blockIdx.x*blockDim.x;

	sdata[tid]=g_idata[i];
	__syncthreads();


	//do reduction in shared memory
	for(unsigned int s=blockDim.x/2;s>0;s>0,s>>=1) {
		if(tid<s) {
			sdata[tid] += sdata[tid+s];
		}
		__syncthreads();
	}

	//write the result for this block to global mem
	if(tid==0)
		g_odata[blockIdx.x]=sdata[0];

}

float reduc3func(int *g_idata,int *g_odata,int GRID_SIZE) {

	GpuTimer gp;
	gp.Start();

	int *h_odata;
	h_odata=(int*)malloc(sizeof(int)*GRID_SIZE);

	reduce3<<<GRID_SIZE,BLOCK_SIZE, sizeof(int)*BLOCK_SIZE>>>(g_idata,g_odata);

	gp.Stop();

	gpuErrchk(hipMemcpy(h_odata,g_odata,sizeof(int)*GRID_SIZE,hipMemcpyDeviceToHost));

//	for(int i=0;i<GRID_SIZE;i++)
//		cout << h_odata[i] << endl;


	int sum=0;
	for(int i=0;i<GRID_SIZE;i++)
			sum+=h_odata[i];

	cout << "Parallel Reduction3 sum = " << sum << endl;

	//cout << gp.Elapsed() << " milli secs (reduction-1)" << endl;

	free(h_odata);

	return gp.Elapsed();
}
	

__global__
void reduce2(int *g_idata,int *g_odata)
{
	extern __shared__ int sdata[];

	//each thread loads one element from global to shared memory
	unsigned int tid=threadIdx.x;
	unsigned int i=threadIdx.x+blockIdx.x*blockDim.x;

	sdata[tid]=g_idata[i];
	__syncthreads();


	//do reduction in shared memory
	for(unsigned int s=1;s<blockDim.x;s*=2) {
		int index = 2*s*tid;

		if(index<blockDim.x) {
			sdata[index] += sdata[index+s];
		}
		__syncthreads();
	}

	//write the result for this block to global mem
	if(tid==0)
		g_odata[blockIdx.x]=sdata[0];

}

float reduc2func(int *g_idata,int *g_odata,int GRID_SIZE) {

	GpuTimer gp;
	gp.Start();

	int *h_odata;
	h_odata=(int*)malloc(sizeof(int)*GRID_SIZE);

	reduce2<<<GRID_SIZE,BLOCK_SIZE, sizeof(int)*BLOCK_SIZE>>>(g_idata,g_odata);

	gp.Stop();

	gpuErrchk(hipMemcpy(h_odata,g_odata,sizeof(int)*GRID_SIZE,hipMemcpyDeviceToHost));

//	for(int i=0;i<GRID_SIZE;i++)
//		cout << h_odata[i] << endl;


	int sum=0;
	for(int i=0;i<GRID_SIZE;i++)
			sum+=h_odata[i];

	cout << "Parallel Reduction2 sum = " << sum << endl;

	//cout << gp.Elapsed() << " milli secs (reduction-1)" << endl;

	free(h_odata);

	return gp.Elapsed();
}
	
__global__
void reduce1(int *g_idata,int *g_odata)
{
	extern __shared__ int sdata[];

	//each thread loads one element from global to shared memory
	unsigned int tid=threadIdx.x;
	unsigned int i=threadIdx.x+blockIdx.x*blockDim.x;

	sdata[tid]=g_idata[i];
	__syncthreads();


	//do reduction in shared memory
	for(unsigned int s=1;s<blockDim.x;s*=2) {
		if((tid%(2*s))==0) {
			sdata[tid] += sdata[tid+s];
		}
		__syncthreads();
	}

	//write the result for this block to global mem
	if(tid==0)
		g_odata[blockIdx.x]=sdata[0];

}

float reduc1func(int *g_idata,int *g_odata,int GRID_SIZE) {

	GpuTimer gp;
	gp.Start();

	int *h_odata;
	h_odata=(int*)malloc(sizeof(int)*GRID_SIZE);

	reduce1<<<GRID_SIZE,BLOCK_SIZE, sizeof(int)*BLOCK_SIZE>>>(g_idata,g_odata);

	gp.Stop();

	gpuErrchk(hipMemcpy(h_odata,g_odata,sizeof(int)*GRID_SIZE,hipMemcpyDeviceToHost));

//	for(int i=0;i<GRID_SIZE;i++)
//		cout << h_odata[i] << endl;


	int sum=0;
	for(int i=0;i<GRID_SIZE;i++)
			sum+=h_odata[i];

	cout << "Parallel Reduction1 sum = " << sum << endl;

	//cout << gp.Elapsed() << " milli secs (reduction-1)" << endl;

	free(h_odata);

	return gp.Elapsed();
}
	

int main() {
	srand(time(0));
	//host allocations
	int *h_idata;

	//allocating and initializing host memory
	h_idata=(int*)malloc(ARRSZ * sizeof(int));
	for(int i=0;i<ARRSZ;i++)
		h_idata[i]=randomNumber();

//	for(int i=0;i<10;i++)
//			cout << h_idata[i] << endl;

	//allocating device memories
	int *g_idata, *g_odata;
	gpuErrchk(hipMalloc((void**)&g_idata,sizeof(int)*ARRSZ));


	int GRID_SIZE=divup(ARRSZ,BLOCK_SIZE);

	cout << "GRID_SIZE= " << GRID_SIZE << endl;
	gpuErrchk(hipMalloc((void**)&g_odata,sizeof(int)*GRID_SIZE));
	//gpuErrchk(hipMemset(g_odata,0,sizeof(int)*GRID_SIZE));

	gpuErrchk(hipMemcpy(g_idata,h_idata,sizeof(int)*ARRSZ,hipMemcpyHostToDevice));

	float elapsedReduc1=reduc1func(g_idata,g_odata,GRID_SIZE);
	float elapsedReduc2=reduc2func(g_idata,g_odata,GRID_SIZE);
	float elapsedReduc3=reduc3func(g_idata,g_odata,GRID_SIZE);
	

	//Serial reduce
	int sum=0;
	for(int i=0;i<ARRSZ;i++)
		sum+=h_idata[i];

	cout << "Serial sum = " << sum << endl;

	cout << "Reduction 1  elapsed time (milli seconds): " << elapsedReduc1 << endl;
	cout << "Reduction 2  elapsed time (milli seconds): " << elapsedReduc2 << endl;
	cout << "Reduction 3  elapsed time (milli seconds): " << elapsedReduc3 << endl;

	cout << "Reduction 1  Effective Bandwidth (GB/s): " << ARRSZ*4*2/elapsedReduc1/1e6 << endl;
	cout << "Reduction 2  Effective Bandwidth (GB/s): " << ARRSZ*4*2/elapsedReduc2/1e6 << endl;
	cout << "Reduction 3  Effective Bandwidth (GB/s): " << ARRSZ*4*2/elapsedReduc3/1e6 << endl;

	free(h_idata);
	hipFree(g_idata);
	hipFree(g_odata);

	return 0;
}
